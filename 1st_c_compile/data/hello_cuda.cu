
#include <hip/hip_runtime.h>
#include <cstddef>
#include <fstream>
#include <iostream>

#define MIN_REDUCE_SYNC_SIZE warpSize

// from Parallel and High Performance Computing, Robert Robey
__device__ void reduction_sum_within_block(int *spad) {
  const unsigned int tiX = threadIdx.x;
  const unsigned int ntX = blockDim.x;

  for (int offset = ntX >> 1; offset > MIN_REDUCE_SYNC_SIZE; offset >>= 1) {
    if (tiX < offset) {
      spad[tiX] = spad[tiX] + spad[tiX + offset];
    }
    __syncthreads();
  }
  if (tiX < MIN_REDUCE_SYNC_SIZE) {
    for (int offset = MIN_REDUCE_SYNC_SIZE; offset > 1; offset >>= 1) {
      spad[tiX] = spad[tiX] + spad[tiX + offset];
      __syncthreads();
    }
    spad[tiX] = spad[tiX] + spad[tiX + 1];
  }
}

__global__ void
reduce_sum_stage1of2(const int isize,  // 0  Total number of cells.
                     const int *array, // 1
                     int *blocksum,    // 2
                     int *redscratch)  // 3
{
  extern __shared__ int spad[];
  const unsigned int giX = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int tiX = threadIdx.x;

  const unsigned int group_id = blockIdx.x;

  spad[tiX] = 0.0;
  if (giX < isize) {
    spad[tiX] = array[giX];
  }

  __syncthreads();

  reduction_sum_within_block(spad);

  //  Write the local value back to an array size of the number of groups
  if (tiX == 0) {
    redscratch[group_id] = spad[0];
    (*blocksum) = spad[0];
  }
}

__global__ void reduce_sum_stage2of2(const int isize, int *total_sum,
                                     int *redscratch) {
  extern __shared__ int spad[];
  const unsigned int tiX = threadIdx.x;
  const unsigned int ntX = blockDim.x;

  int giX = tiX;

  spad[tiX] = 0.0;

  // load the sum from reduction scratch, redscratch
  if (tiX < isize)
    spad[tiX] = redscratch[giX];

  for (giX += ntX; giX < isize; giX += ntX) {
    spad[tiX] += redscratch[giX];
  }

  __syncthreads();

  reduction_sum_within_block(spad);

  if (tiX == 0) {
    (*total_sum) = spad[0];
  }
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    std::cout << "Usage: " << argv[0] << " <input_file>" << std::endl;
    return 1;
  }

  std::ifstream input_file(argv[1], std::ios::binary);
  int n;
  input_file.read(reinterpret_cast<char *>(&n), sizeof(n));
  int *data = new int[n];
  input_file.read(reinterpret_cast<char *>(data), n * sizeof(int));

  size_t blocksize = 128;
  size_t blocksizebytes = blocksize * sizeof(int);
  size_t global_work_size = ((n + blocksize - 1) / blocksize) * blocksize;
  size_t gridsize = global_work_size / blocksize;

  int *device_data, *device_sum, *device_redscratch;
  hipMalloc(&device_data, n * sizeof(int));
  hipMalloc(&device_sum, sizeof(int));
  hipMalloc(&device_redscratch, gridsize * sizeof(int));

  hipMemcpy(device_data, data, n * sizeof(int), hipMemcpyHostToDevice);

  reduce_sum_stage1of2<<<gridsize, blocksize, blocksizebytes>>>(
      n, device_data, device_sum, device_redscratch);

  if (gridsize > 1) {
    reduce_sum_stage2of2<<<1, blocksize, blocksizebytes>>>(n, device_sum,
                                                           device_redscratch);
  }

  int sum;
  hipMemcpy(&sum, device_sum, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "cuda" << std::endl;
  std::cout << sum << std::endl;
  
  hipFree(device_redscratch);
  hipFree(device_sum);
  hipFree(device_data);

  delete[] data;
  return 0;
}

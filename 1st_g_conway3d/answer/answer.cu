#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <chrono>
#include <iostream>



template <int size>
__global__ void life(uint32_t* datain, uint32_t* dataout, int xs, int ys, int zs) {
    int y = blockIdx.z;

    __shared__ uint32_t tempsrc[(size / 4) * 4];
    __shared__ uint32_t tempin[(size / 4 + 2) * 6];
    uint8_t* tempsrc_ = (uint8_t*)tempsrc;
    //uint8_t* tempin_ = (uint8_t*)tempin;
    uint32_t tempout[4];

    int z = blockIdx.y;

    int z_[3];
    z_[0] = (z + zs - 1) % zs;
    z_[1] = z;
    z_[2] = (z + 1) % zs;

    uint32_t loc[6];
    for (int j = 0; j < 6; j++) {
        int y_ = y * 4 + ys - 1 + j;
        y_ = y_ % ys;

        loc[j] = 0;
        for (int k = 0; k < 3; k++) {
            int i_ = (z_[k] * ys + y_) * xs + threadIdx.x;

            loc[j] += datain[i_];
        }
    }


    for (int j = 0; j < 4; j++) {
        int y_ = y * 4 + j;

        int i_ = (z * ys + y_) * xs + threadIdx.x;
        uint32_t loc = datain[i_];

        int ini = j * (size / 4) + threadIdx.x;
        tempsrc[ini] = loc;
    }
    __syncthreads();


    loc[0] = loc[0] + loc[1] + loc[2];
    loc[1] = loc[1] + loc[2] + loc[3];
    loc[2] = loc[2] + loc[3] + loc[4];
    loc[3] = loc[3] + loc[4] + loc[5];

    tempin[0 * (size / 4 + 2) + threadIdx.x + 1] = loc[0];
    tempin[1 * (size / 4 + 2) + threadIdx.x + 1] = loc[1];
    tempin[2 * (size / 4 + 2) + threadIdx.x + 1] = loc[2];
    tempin[3 * (size / 4 + 2) + threadIdx.x + 1] = loc[3];
    __syncthreads();


    if (threadIdx.x < 4) {
        tempin[threadIdx.x * (size / 4 + 2) + 0] = tempin[threadIdx.x * (size / 4 + 2) + (size / 4)];
        tempin[threadIdx.x * (size / 4 + 2) + (size / 4 + 1)] = tempin[threadIdx.x * (size / 4 + 2) + 1];
    }
    __syncthreads();


    for (int j = 0; j < 4; j++) {
        uint32_t loc0 = tempin[j * (size / 4 + 2) + threadIdx.x];
        uint32_t loc1 = tempin[j * (size / 4 + 2) + threadIdx.x + 1];
        uint32_t loc2 = tempin[j * (size / 4 + 2) + threadIdx.x + 2];
        loc0 = loc1 + (loc1 >> 8) + (loc2 << 24) + (loc1 << 8) + (loc0 >> 24);


        for (int step = 0; step < 4; step++) {

            uint32_t c = loc0 & 0xff;
            loc0 = loc0 >> 8;

            uint32_t loc = tempsrc_[(j * (size / 4) + threadIdx.x) * 4 + step];
            if (c == 6 || 6 <= c && c <= 8 && loc) {
                tempout[step] = 1;
            }
            else {
                tempout[step] = 0;
            }
        }

        uint32_t out = tempout[0] + (tempout[1] << 8) + (tempout[2] << 16) + (tempout[3] << 24);
        int y_ = y * 4 + j;
        int i_ = (z * ys + y_) * xs + threadIdx.x;
        dataout[i_] = out;
    }
}


void tofile(void* p, size_t n, const char* fn) {
    FILE* fi = fopen(fn, "wb");
    fwrite(p, 1, n, fi);
    fclose(fi);
}

void fromfile(void* p, size_t n, const char* fn) {
    FILE* fi = fopen(fn, "rb");
    fread(p, 1, n, fi);
    fclose(fi);
}

int main(int argc, const char** argv)
{
    if (argc < 4) {
        std::cout << "Usage: " << argv[0] << " <input_path> <output_path> <N>" << std::endl;

        argv = (const char**)malloc(32);
        argv[1] = "../../conf.data";
        argv[2] = "../../out.data";
        argv[3] = "2";
    }

    int itn;
    sscanf(argv[3], "%d", &itn);

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
    }

    uint32_t* a;
    uint32_t* dev_a = 0;
    uint32_t* dev_b = 0;

    int64_t size;
    int64_t t;

    FILE* fi = fopen(argv[1], "rb");
    fread(&size, 1, 8, fi);
    fread(&t, 1, 8, fi);
    a = (uint32_t*)malloc(size * size * size);
    fread(a, 1, size * size * size, fi);
    fclose(fi);

    cudaStatus = hipMalloc((void**)&dev_a, size * size * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * size * size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(dev_a, a, size * size * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "copy failed!");
    }

    dim3 g = dim3(1, size, size / 4);
    dim3 b = dim3(size / 4, 1, 1);

    auto t1 = std::chrono::steady_clock::now();

    if (size == 256) {
        for (int i = 0; i < itn / 2; i++) {
            life<256> << <g, b >> > (dev_a, dev_b, size / 4, size, size);
            life<256> << <g, b >> > (dev_b, dev_a, size / 4, size, size);
        }
    }
    else if (size == 512) {
        for (int i = 0; i < itn / 2; i++) {
            life<512> << <g, b >> > (dev_a, dev_b, size / 4, size, size);
            life<512> << <g, b >> > (dev_b, dev_a, size / 4, size, size);
        }
    }
    else if (size == 1024) {
        for (int i = 0; i < itn / 2; i++) {
            life<1024> << <g, b >> > (dev_a, dev_b, size / 4, size, size);
            life<1024> << <g, b >> > (dev_b, dev_a, size / 4, size, size);
        }
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calc failed");
    }
    auto t2 = std::chrono::steady_clock::now();

    cudaStatus = hipMemcpy(a, dev_a, size * size * size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    t += itn;
    fi = fopen(argv[2], "wb");
    fwrite(&size, 1, 8, fi);
    fwrite(&t, 1, 8, fi);
    fwrite(a, 1, size * size * size, fi);
    fclose(fi);

    int d1 = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%d\n", d1);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    return 0;
}
